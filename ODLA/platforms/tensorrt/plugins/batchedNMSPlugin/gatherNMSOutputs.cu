#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
 
//===- gatherNMSOutputs.cu ------------------------------------------------===//
//
// Copyright (C) 2019-2020 Alibaba Group Holding Limited.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// =============================================================================

#include "gatherNMSOutputs.h"
#include "kernel.h"
#include "plugin.h"
#include <vector>

template <typename T_BBOX, typename T_SCORE, unsigned nthds_per_cta>
__launch_bounds__(nthds_per_cta) __global__ void gatherNMSOutputs_kernel(const bool shareLocation, const int numImages,
    const int numPredsPerClass, const int numClasses, const int topK, const int keepTopK, const int* indices,
    const T_SCORE* scores, const T_BBOX* bboxData, int* numDetections, T_BBOX* nmsedBoxes, T_BBOX* nmsedScores,
    T_BBOX* nmsedClasses, int* nmsedIndices, bool clipBoxes)
{
    if (keepTopK > topK)
        return;
    for (int i = blockIdx.x * nthds_per_cta + threadIdx.x; i < numImages * keepTopK; i += gridDim.x * nthds_per_cta)
    {
        const int imgId = i / keepTopK;
        const int detId = i % keepTopK;
        const int offset = imgId * numClasses * topK;
        const int index = indices[offset + detId];
        const T_SCORE score = scores[offset + detId];
        if (index == -1)
        {
            nmsedClasses[i] = -1;
            nmsedScores[i] = 0;
            nmsedBoxes[i * 4] = 0;
            nmsedBoxes[i * 4 + 1] = 0;
            nmsedBoxes[i * 4 + 2] = 0;
            nmsedBoxes[i * 4 + 3] = 0;
            nmsedIndices[i * 3] = -1;
            nmsedIndices[i * 3 + 1] = -1;
            nmsedIndices[i * 3 + 2] = -1;
        }
        else
        {
            const int bboxOffset = imgId * (shareLocation ? numPredsPerClass : (numClasses * numPredsPerClass));
            const int bboxId
                = ((shareLocation ? (index % numPredsPerClass) : index % (numClasses * numPredsPerClass)) + bboxOffset)
                * 4;
            nmsedClasses[i] = (index % (numClasses * numPredsPerClass)) / numPredsPerClass; // label
            nmsedScores[i] = score;                                                         // confidence score
            // clipped bbox xmin
            nmsedBoxes[i * 4] = clipBoxes ? max(min(bboxData[bboxId], T_BBOX(1.)), T_BBOX(0.)) : bboxData[bboxId];
            // clipped bbox ymin
            nmsedBoxes[i * 4 + 1]
                = clipBoxes ? max(min(bboxData[bboxId + 1], T_BBOX(1.)), T_BBOX(0.)) : bboxData[bboxId + 1];
            // clipped bbox xmax
            nmsedBoxes[i * 4 + 2]
                = clipBoxes ? max(min(bboxData[bboxId + 2], T_BBOX(1.)), T_BBOX(0.)) : bboxData[bboxId + 2];
            // clipped bbox ymax
            nmsedBoxes[i * 4 + 3]
                = clipBoxes ? max(min(bboxData[bboxId + 3], T_BBOX(1.)), T_BBOX(0.)) : bboxData[bboxId + 3];
            nmsedIndices[i * 3] = imgId;
            nmsedIndices[i * 3 + 1] = (int) nmsedClasses[i];
            nmsedIndices[i * 3 + 2] = bboxId / 4;
            atomicAdd(&numDetections[i / keepTopK], 1);
        }
    }
}

template <typename T_BBOX, typename T_SCORE>
pluginStatus_t gatherNMSOutputs_gpu(hipStream_t stream, const bool shareLocation, const int numImages,
    const int numPredsPerClass, const int numClasses, const int topK, const int keepTopK, const void* indices,
    const void* scores, const void* bboxData, void* numDetections, void* nmsedBoxes, void* nmsedScores,
    void* nmsedClasses, void* nmsedIndices, bool clipBoxes)
{
    hipMemsetAsync(numDetections, 0, numImages * sizeof(int), stream);
    const int BS = 32;
    const int GS = 32;
    gatherNMSOutputs_kernel<T_BBOX, T_SCORE, BS><<<GS, BS, 0, stream>>>(shareLocation, numImages, numPredsPerClass,
        numClasses, topK, keepTopK, (int*) indices, (T_SCORE*) scores, (T_BBOX*) bboxData, (int*) numDetections,
        (T_BBOX*) nmsedBoxes, (T_BBOX*) nmsedScores, (T_BBOX*) nmsedClasses, (int*) nmsedIndices, clipBoxes);

    CSC(hipGetLastError(), STATUS_FAILURE);
    return STATUS_SUCCESS;
}

// gatherNMSOutputs LAUNCH CONFIG {{{
typedef pluginStatus_t (*nmsOutFunc)(hipStream_t, const bool, const int, const int, const int, const int, const int,
    const void*, const void*, const void*, void*, void*, void*, void*, void*, bool);
struct nmsOutLaunchConfig
{
    DataType t_bbox;
    DataType t_score;
    nmsOutFunc function;

    nmsOutLaunchConfig(DataType t_bbox, DataType t_score)
        : t_bbox(t_bbox)
        , t_score(t_score)
    {
    }
    nmsOutLaunchConfig(DataType t_bbox, DataType t_score, nmsOutFunc function)
        : t_bbox(t_bbox)
        , t_score(t_score)
        , function(function)
    {
    }
    bool operator==(const nmsOutLaunchConfig& other)
    {
        return t_bbox == other.t_bbox && t_score == other.t_score;
    }
};

using nvinfer1::DataType;

static std::vector<nmsOutLaunchConfig> nmsOutFuncVec;

bool nmsOutputInit()
{
    nmsOutFuncVec.push_back(nmsOutLaunchConfig(DataType::kFLOAT, DataType::kFLOAT, gatherNMSOutputs_gpu<float, float>));
    return true;
}

static bool initialized = nmsOutputInit();

//}}}

pluginStatus_t gatherNMSOutputs(hipStream_t stream, const bool shareLocation, const int numImages,
    const int numPredsPerClass, const int numClasses, const int topK, const int keepTopK, const DataType DT_BBOX,
    const DataType DT_SCORE, const void* indices, const void* scores, const void* bboxData, void* numDetections,
    void* nmsedBoxes, void* nmsedScores, void* nmsedClasses, void* nmsedIndices, bool clipBoxes)
{
    nmsOutLaunchConfig lc = nmsOutLaunchConfig(DT_BBOX, DT_SCORE);
    for (unsigned i = 0; i < nmsOutFuncVec.size(); ++i)
    {
        if (lc == nmsOutFuncVec[i])
        {
            DEBUG_PRINTF("gatherNMSOutputs kernel %d\n", i);
            return nmsOutFuncVec[i].function(stream, shareLocation, numImages, numPredsPerClass, numClasses, topK,
                keepTopK, indices, scores, bboxData, numDetections, nmsedBoxes, nmsedScores, nmsedClasses, nmsedIndices,
                clipBoxes);
        }
    }
    return STATUS_BAD_PARAM;
}